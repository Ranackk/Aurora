#include "hip/hip_runtime.h"
#include "stdafx.h"

#include "Rendering/Scenes/SceneHyperPlayground.h"
#include "Rendering/Scenes/SceneTypes.h"
#include "Rendering/Scenes/SDFFactory.h"

#include "Options/Configuration.h"

#include "MathLib/MathLib.h"
#include "../Application.h"

//////////////////////////////////////////////////////////////////////////

void SceneHyperPlayground::Update(Configuration& config) {	
	auto time = Application::s_Instance->GetTimeSinceStartupMyS();
	for (int i = 0; i < 6; i++)
	{
		if (config.SceneAnimateRotations[i]) 
		{
			config.SceneSliderRotations[i] = fmod((time.count() / 1000000.0f * config.SceneSpeed), glm::two_pi<float>());
		}
	}

	glm::mat4 transformation = Math::RotZW(config.SceneSliderRotations[0]) * 
							   Math::RotYW(config.SceneSliderRotations[1]) * 
							   Math::RotYZ(config.SceneSliderRotations[2]) * 
							   Math::RotXW(config.SceneSliderRotations[3]) * 
							   Math::RotXZ(config.SceneSliderRotations[4]) *
							   Math::RotXY(config.SceneSliderRotations[5]);

	glm::vec4 translation	= {config.SceneSliderPositions[0], config.SceneSliderPositions[1], config.SceneSliderPositions[2], config.SceneSliderPositions[3]};

	m_SDF_Cube->GetSDF().SetTransformationMatrix(transformation);
	m_SDF_Cube->SetTranslationVector(m_BaseTranslation + translation);
}

//////////////////////////////////////////////////////////////////////////

A_CUDA_CPUGPU float SceneHyperPlayground::EvaluateDistance(const glm::vec4& position) const
{
	return m_SDF_Cube->EvaluateDistance(position);
}

//////////////////////////////////////////////////////////////////////////

A_CUDA_CPUGPU glm::vec4 SceneHyperPlayground::EvaluateNormal(const glm::vec4& position) const
{
	return Math::SampleNormal(*m_SDF_Cube, position);
}

//////////////////////////////////////////////////////////////////////////

A_CUDA_CPUGPU glm::vec4 SceneHyperPlayground::GetLocalSamplePosition(const glm::vec4& position) const
{
	return m_SDF_Cube->GetLocalSamplePosition(position);
}

//////////////////////////////////////////////////////////////////////////

A_CUDA_CPUGPU glm::vec4 SceneHyperPlayground::EvaluateToSurfaceVectorZW(const glm::vec4& position, float& outDistance) const
{
	return Math::EvaluateToSurfaceVectorZW(*m_SDF_Cube, position, outDistance);
}

//////////////////////////////////////////////////////////////////////////

A_CUDA_CPUGPU glm::vec4 SceneHyperPlayground::EvaluateToSurfaceVector(const glm::vec4& position, float& outDistance) const
{
	return Math::EvaluateToSurfaceVector(*m_SDF_Cube, position, outDistance);
}

//////////////////////////////////////////////////////////////////////////

void SceneHyperPlayground::Init()
{
	m_SDF				= SDFFactory::CreateSDF_HyperCube();
	m_SDF_Plane			= m_SDF->GetSDF().GetRHS();
	m_SDF_Cube			= m_SDF->GetSDF().GetLHS();

	m_BaseTranslation	= m_SDF_Cube->GetTranslation();
}

//////////////////////////////////////////////////////////////////////////

void SceneHyperPlayground::UnInit()
{
	delete m_SDF;
}
