#include "hip/hip_runtime.h"
 #pragma once

#include <hip/hip_runtime.h>
#include <>
#include <cstdio>

#include "GraphicsIncludes.h"

#include "Marching/MarchingTypes.h"
#include "Marching/MarchingFunctions.h"
#include "MathLib/MathLib.h"

#include "Rendering/CUDATypes.h"
#include "Rendering/CUDAInterface.h"
#include "Rendering/Camera.h"
#include "Rendering/Light.h"

#include "Options/Configuration.h"


#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

constexpr unsigned int BLOCK_SIZE_2D = 16;
//constexpr unsigned int BLOCK_SIZE_3D = 8;
constexpr unsigned int RESULT_COLOR_COMPONENT_COUNT = 4;

////////////////////////////////////////////////////////////////

using N = glm::vec4;

A_CUDA_KERNEL void k_RenderPixel(RenderPixelBufferDataCUDA* bufferData, RenderSceneDataCUDA* sceneData, Configuration* config, Camera<glm::vec4>* camera, Light<glm::vec4>* light);

void CUDA_RenderImage(RenderPixelBufferDataCUDA* bufferData, RenderSceneDataCUDA* sceneData, Configuration* config, Camera<glm::vec4>* camera, Light<glm::vec4>* light);

void CUDA_PrepareRenderImage(RenderingBuffer& RenderingBuffer, hipSurfaceObject_t& outSurfaceObject);
void CUDA_FinishRenderImage(RenderingBuffer& RenderingBuffer);

////////////////////////////////////////////////////////////////
// Slice Rendering (4D -> 2D)
////////////////////////////////////////////////////////////////

// Needs to be called form OpenGL Thread
void CUDA_PrepareRenderImage(RenderingBuffer& RenderingBuffer, hipSurfaceObject_t& outSurfaceObject)
{
	if (!RenderingBuffer.IsCurrentlyMapped)
	{
		CUDA_CHECK_ERROR(hipGraphicsMapResources(1, &RenderingBuffer.d_CUDAGraphicsResource));
		RenderingBuffer.IsCurrentlyMapped = true;
	}
	
	hipArray_t arrayDPtr;
	CUDA_CHECK_ERROR(hipGraphicsSubResourceGetMappedArray(&arrayDPtr, RenderingBuffer.d_CUDAGraphicsResource, 0, 0));

	// Create the cuda resource description
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType			= hipResourceTypeArray;    // be sure to set the resource type to hipResourceTypeArray
	resDesc.res.array.array = arrayDPtr;				// this is the important bit
 
	// Create the surface object
	CUDA_CHECK_ERROR(hipCreateSurfaceObject(&outSurfaceObject, &resDesc));
}

// Needs to be called form OpenGL Thread
void CUDA_FinishRenderImage(RenderingBuffer& RenderingBuffer)
{
	if (RenderingBuffer.IsCurrentlyMapped)
	{
		CUDA_CHECK_ERROR(hipGraphicsUnmapResources(1, &RenderingBuffer.d_CUDAGraphicsResource));
		RenderingBuffer.IsCurrentlyMapped = false;
	}
}

// May be called from any Thread
void CUDA_RenderImage(RenderPixelBufferDataCUDA* bufferData, RenderSceneDataCUDA* sceneData, Configuration* config, Camera<glm::vec4>* camera, Light<glm::vec4>* light)
{	
	CUDA_CHECK_ERROR(hipGetLastError());
	
	////////////////////////////////////////////////////////////////

	//printf("Start Render Image\n");
	const dim3 threadsPerBlock	= dim3(BLOCK_SIZE_2D, BLOCK_SIZE_2D);
	const dim3 numBlocks		= dim3(bufferData->BufferDimensions.x / BLOCK_SIZE_2D, bufferData->BufferDimensions.y /BLOCK_SIZE_2D);

	constexpr bool SHOW_DEBUG = false;
	if (SHOW_DEBUG)
	{
		// 32 * 32 = 1024;
		struct hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, 0);
		printf("using %i multiprocessors\n", properties.multiProcessorCount);
		printf("max threads per processor: %i\n", properties.maxThreadsPerMultiProcessor);
		printf("params: threadsPerBlock (%i, %i), numBlocks (%i, %i) \n", threadsPerBlock.x, threadsPerBlock.y, numBlocks.x, numBlocks.y);
	}

	hipDeviceSynchronize();
	CUDA_CHECK_ERROR(hipGetLastError());

	k_RenderPixel KERNEL_ARGS2(numBlocks, threadsPerBlock)(bufferData, sceneData, config, camera, light);
	
	hipDeviceSynchronize();
	CUDA_CHECK_ERROR(hipGetLastError());
	
	////////////////////////////////////////////////////////////////

	//printf("End Render Image\n");
  }

////////////////////////////////////////////////////////////////

A_CUDA_KERNEL void k_RenderPixel(RenderPixelBufferDataCUDA* bufferData, RenderSceneDataCUDA* sceneData, Configuration* config, Camera<glm::vec4>* camera, Light<glm::vec4>* light)
{
	#define USE_BIRAY_MARCHING

	// Global

	const int pixelX		= blockIdx.x * blockDim.x + threadIdx.x;
	const int pixelY		= blockIdx.y * blockDim.y + threadIdx.y;
	
	const int viewX			= pixelX / bufferData->ViewDimensions.x; 
	const int viewY			= pixelY / bufferData->ViewDimensions.y;
	const int viewID		= viewY * bufferData->NumViews.x + viewX;
	const int viewCount		= bufferData->NumViews.x * bufferData->NumViews.y;

	const int viewOriginX	= viewX * bufferData->ViewDimensions.x;
	const int viewOriginY	= viewY * bufferData->ViewDimensions.y;

	const float viewPercentage	= (viewCount == 1) ? 0.5f : viewID / static_cast<float>(viewCount);

	// In View

	const int inViewX		= pixelX - viewOriginX;
	const int inViewY		= pixelY - viewOriginY;
	
	// Ray
	const float inViewPercentageX = inViewX / static_cast<float>(bufferData->ViewDimensions.x);
	const float inViewPercentageY = inViewY / static_cast<float>(bufferData->ViewDimensions.y);
	
	constexpr float SCISSOR_RECT_SIZE_X			= 0.40f;
	constexpr float SCISSOR_RECT_SIZE_X_HALF	= SCISSOR_RECT_SIZE_X / 2.0f;
	constexpr float SCISSOR_RECT_SIZE_Y			= 0.60f;
	constexpr float SCISSOR_RECT_SIZE_Y_HALF	= SCISSOR_RECT_SIZE_Y / 2.0f;
	constexpr float GROUND_PLANE_Y				= 0.0f;

	const bool isInGroundPlane = inViewPercentageY < GROUND_PLANE_Y;
	const bool isInScissorRect = inViewPercentageX > (0.5f - SCISSOR_RECT_SIZE_X_HALF) && inViewPercentageX < (0.5f + SCISSOR_RECT_SIZE_X_HALF) &&
		 				   inViewPercentageY > (0.5f - SCISSOR_RECT_SIZE_Y_HALF) && inViewPercentageY < (0.5f + SCISSOR_RECT_SIZE_Y_HALF);
	
	// March Ray

	RayMarchResult<glm::vec4> result;
	if (isInGroundPlane)
	{
		// Render Ground Plane via Raytracing

		// Calculate intersection point between ray and plane. Note: We do only use a ray here, not a biray.
		glm::highp_mat4 biRaySpaceToWorldSpace;
		const Math::BiRay<glm::vec4> biRay	= camera->GetBiray(viewPercentage, inViewPercentageX, inViewPercentageY, biRaySpaceToWorldSpace);

		constexpr float GROUND_POSITION_Y	= -100.0f;
		const float traversedMain			= GROUND_POSITION_Y - biRay.Origin.y / biRay.DirectionMain.y;
		const glm::vec4 position			= biRay.At(traversedMain, 0);
		const glm::vec4 normal				= glm::vec4(0, 1, 0, 0);

		result = RayMarchResult<glm::vec4>(true, traversedMain, 1, 0, 0, position, position, position, normal, normal);
	}
	else if (isInScissorRect)
	{
		// Render Scene	via WaveMarching	

		glm::highp_mat4 biRaySpaceToWorldSpace;
		const Math::BiRay<glm::vec4> biRay	= camera->GetBiray(viewPercentage, inViewPercentageX, inViewPercentageY, biRaySpaceToWorldSpace);
		result								= RayMarchFunctions::MarchSingleBiRay<glm::vec4, glm::mat4>(biRay, biRaySpaceToWorldSpace, sceneData, config->MIN_STEP_SIZE, config->MAX_DEPTH, config->MAX_STEPS, config->RAY_HIT_EPSILON);
	}
	else
	{
		// Neither the main scene nor the ground plane is rendered, so we do not alter the "not hit" result.
		result = RayMarchResult<glm::vec4>();
		result.Hit = false;
	}

	// Soft shadows
	if (result.Hit)
	{
		const glm::vec4 toLightPosition		= light->Position - result.Position;
		const float toLightDistance			= glm::length(toLightPosition);
		const glm::vec4 toLightPositionN	= toLightPosition / toLightDistance;

		// Shadow Ray
		const Math::Ray<glm::vec4> shadowRay = Math::Ray<glm::vec4>(result.Position + toLightPositionN * config->SHADOW_START_OFFSET, toLightPositionN);
		result.ShadowValue					 = RayMarchFunctions::MarchSecondaryShadowRay<glm::vec4>(shadowRay, sceneData, toLightDistance, light->Radius, config->MAX_STEPS_SHADOW, config->RAY_HIT_EPSILON, config->SHADOW_PENUMBRA);
	}

	// Color in
	const uchar4 color = isInGroundPlane ? VisualizationHelper::GetColorForRayResult_SimpleLit(*config, result) : VisualizationHelper::GetColorForRayResult(*config, result);
	surf2Dwrite(color, bufferData->SurfaceObject, RESULT_COLOR_COMPONENT_COUNT * sizeof(BufferType) * pixelX, pixelY);
}